#include <iostream>
#include <fstream>

#include "../h/MT.h"

#include "../h/particles.cuh"
#include "../h/parameters.cuh"

uint IDs;
uint IDe;
float tmax;
float Tfin;

using namespace PhysPeach;
int main(){
    std::cout << "hello, test" << std::endl;

    //test
    Tfin = 1;
    tmax = 1;
    IDs = 0;
    IDe = 0;

    //initialise random func
    init_genrand((unsigned long)time(NULL));

    Particles p;
    makeParticles(&p);
    scatterParticles(&p, 10.0);

    //test initParticles
    std::ofstream checkScatterPositions("testData/checkScatterParticles.data");
    
    hipMemcpy(p.diam, p.diam_dev, NP * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(p.x, p.x_dev, D * NP * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(p.v, p.v_dev, D * NP * sizeof(float), hipMemcpyDeviceToHost);

    for(uint n = 0; n < NP; n++){
        checkScatterPositions << p.diam[n] << " ";
    }
    checkScatterPositions << std::endl;

    for(uint n = 0; n < NP; n++){
        for(char d = 0; d < D; d++){
            checkScatterPositions << p.x[d * NP + n] << " ";
        }
        for(char d = 0; d < D; d++){
            checkScatterPositions << p.v[d * NP + n] << " ";
        }
        checkScatterPositions << std::endl;
    }

    checkScatterPositions.close();
    
    killParticles(&p);

    std::cout << "makeParticles done!" << std::endl;
    return 0;
}