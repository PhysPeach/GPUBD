#include <iostream>
#include <fstream>

#include "../h/MT.h"

#include "../h/particles.cuh"
#include "../h/parameters.cuh"

uint IT;
uint IDs;
uint IDe;
float tau;
float Tfin;

using namespace PhysPeach;
int main(){
    std::cout << "hello, test" << std::endl;

    //test
    Tfin = 1;
    tau = 100;
    IDs = 1;
    IDe = 1;

    //initialise random func
    init_genrand((unsigned long)time(NULL));

    std::cout << "---Settings---" << std::endl;
    std::cout << "Tfin = " << Tfin << std::endl;
    std::cout << "t_eq = " << tau << std::endl;
    std::cout << "t_rec = " << tau << std::endl;
    std::cout << "ID = [" << IDs << ", " << IDe << "]" << std::endl;
    std::cout << "--------------" << std::endl;

    Particles p;
    makeParticles(&p);
    initParticles(&p, 10.0);

    //test initParticles
    std::ofstream checkInitPositions("testData/checkInitParticles.data");
    
    hipMemcpy(p.diam, p.diam_dev, NP * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(p.x, p.x_dev, D * NP * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(p.v, p.v_dev, D * NP * sizeof(float), hipMemcpyDeviceToHost);

    for(uint n = 0; n < NP; n++){
        checkInitPositions << p.diam[n] << " ";
    }
    checkInitPositions << std::endl;

    for(uint n = 0; n < NP; n++){
        for(char d = 0; d < D; d++){
            checkInitPositions << p.x[d * NP + n] << " ";
        }
        for(char d = 0; d < D; d++){
            checkInitPositions << p.v[d * NP + n] << " ";
        }
        checkInitPositions << std::endl;
    }

    checkInitPositions.close();
    
    killParticles(&p);

    std::cout << "makeParticles done!" << std::endl;
    return 0;
}