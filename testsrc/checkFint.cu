#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include "../h/MT.h"

#include "../h/box.cuh"
#include "../h/parameters.cuh"

uint IDs;
uint IDe;
float tau;
float Tfin;

using namespace PhysPeach;
int main(){
    std::cout << "hello, test" << std::endl;

    //test
    Tfin = 1;
    tau = 100;
    IDs = 0;
    IDe = 0;

    //initialise random func
    init_genrand((unsigned long)time(NULL));

    std::cout << "---Settings---" << std::endl;
    std::cout << "Tfin = " << Tfin << std::endl;
    std::cout << "t_eq = " << tau << std::endl;
    std::cout << "t_rec = " << tau << std::endl;
    std::cout << "ID = [" << IDs << ", " << IDe << "]" << std::endl;
    std::cout << "--------------" << std::endl;

    float force[D*NP];

    Box box;
    makeBox(&box);
    initBox(&box, 1);

    float av = 0;
    float sig = 0.;
    hipMemcpy(force, box.p.force_dev, D*NP*sizeof(float),hipMemcpyDeviceToHost);
    for(uint i = 0; i < D*NP; i++){
        std::cout << i << ": " << force[i] << std::endl;
        av += force[i]/(D*NP);
    }
    for(uint i = 0; i < D*NP; i++){
        sig += (av - force[i])*(av - force[i])/(D*NP);
    }
    sig = sqrt(sig);
    std::cout <<"force: av = " << av << ", sig = " << sig << std::endl;

    killBox(&box);

    return 0;
}