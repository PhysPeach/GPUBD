#include "../h/grid.cuh"

namespace PhysPeach{
    void makeGrid(Grid* grid, float L){
        ////define M ~ L/Rcell: Rcell ~ 5a
        grid->M = (uint)(L/(4.8*a0));
        ////for small system
        if(grid->M < 3){
            grid->M = 3;
        }
        grid->rc = L/(float)grid->M;
        uint M2 = grid->M * grid->M;
        grid->EpM = (uint)(1.5 * (float)NP /(float)M2); //EpM ~ NP/M^D

        hipMalloc((void**)&grid->cell_dev, M2 * grid->EpM * sizeof(uint));

        //for parallel interactions
        IT = grid->EpM * NG * NG;
        uint M_NG = grid->M/NG + 0.9;
        hipMalloc((void**)&grid->refCell_dev, M_NG * M_NG * sizeof(uint));

        return;
    }
    void killGrid(Grid* grid){
        hipFree(grid->refCell_dev);
        hipFree(grid->cell_dev);
    }
}