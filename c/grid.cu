#include "hip/hip_runtime.h"
#include "../h/grid.cuh"

namespace PhysPeach{
    void makeGrid(Grid* grid, float L){
        //define M ~ L/Rcell: Rcell ~ 5a
        grid->M = (uint)(L/(4.8*a0));
        //for small system
        if(grid->M < 3){
            grid->M = 3;
        }
        grid->rc = L/(float)grid->M;
        uint M2 = grid->M * grid->M;
        grid->EpM = (uint)(1.5 * (float)NP /(float)M2); //EpM ~ NP/M^D

        hipMalloc((void**)&grid->cell_dev, M2 * grid->EpM * sizeof(uint));

        //for parallel interactions
        IT = grid->EpM * NG * NG;
        uint M_NG = grid->M/NG + 0.9;
        hipMalloc((void**)&grid->refCell_dev, M_NG * M_NG * sizeof(uint));
        makeCellPattern2D(grid);

        return;
    }
    void killGrid(Grid* grid){
        hipFree(grid->refCell_dev);
        hipFree(grid->cell_dev);
        return;
    }
    void makeCellPattern2D(Grid* grid){
        uint M_NG = grid->M/NG + 0.9;
        uint pattern[M_NG*M_NG];

        uint foo = 0;
        uint hoge = 0;
        uint hhoge = 0;
        for(uint m_NG2 = 0; m_NG2 < M_NG*M_NG; m_NG2++){
            hhoge = m_NG2/M_NG;
            hoge = m_NG2 - hhoge * M_NG;
            if(!hoge){
                foo = NG * hhoge * grid->M;
            }
            pattern[m_NG2] = foo;
            foo += NG;
        }
        hipMemcpy(grid->refCell_dev, pattern, M_NG * M_NG * sizeof(uint), hipMemcpyHostToDevice);
        return;
    }
    __global__ void updateGrid2D(Grid grid, uint* cell, float* x){
        uint n_global = blockIdx.x * blockDim.x + threadIdx.x;

        uint M = grid.M;
        uint EpM = grid.EpM;
        float rc = grid.rc;
    
        uint cellPos[D];
        uint cellAddress;//[0, M * M - 1]
        uint n_m;
        uint counter;

        for(uint n = n_global; n < NP; n += NB * NT){
            cellPos[0] = (uint)(x[n]/rc);
            cellPos[1] = (uint)(x[NP+n]/rc);
            cellAddress = cellPos[1] * M + cellPos[0];
            n_m = cellAddress * EpM;
            counter = 1 + atomicAdd(&cell[n_m], 1);
            cell[n_m + counter] = n;
        }
    }
}