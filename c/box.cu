#include "hip/hip_runtime.h"
#include "../h/box.cuh"

namespace PhysPeach{
    void makeGridPattern2D(Box* box){
        uint M_NG = box->M/NG + 0.9;
        uint pattern[M_NG*M_NG];

        uint foo = 0;
        uint hoge = 0;
        uint hhoge = 0;
        for(uint m_NG2 = 0; m_NG2 < M_NG*M_NG; m_NG2++){
            hhoge = m_NG2/M_NG;
            hoge = m_NG2 - hhoge * M_NG;
            if(!hoge){
                foo = NG * hhoge * box->M;
            }
            pattern[m_NG2] = foo;
            foo += NG;
        }
        hipMemcpy(box->refGrid_dev, pattern, M_NG * M_NG * sizeof(uint), hipMemcpyHostToDevice);
        return;
    }
    void makeBox(Box* box){

        makeParticles(&box->p);

        //default settings
        box->id = 0;
        box->dt = dt_MD;
        box->t = 0.;
        box->T = Tfin;
        box->L = sqrt(NP/DNSTY);
        box->thermalFuctor = sqrt(2*box->T/box->dt);

        //for list
        ////define M ~ L/Rcell: Rcell ~ 4~5a
        box->M = (uint)(box->L/(4.3*a0));
        ////for small system
        if(box->M < 3){
            box->M = 3;
        }
        uint M2 = box->M * box->M;
        box->EpM = (uint)1.8 * DNSTY * (4.3*a0)*(4.3*a0); //EpM ~ DNSTY * 4.3a0^2 ~ DNSTY * 26.6
        hipMalloc((void**)&box->needUpdate_dev, sizeof(uint));
        hipMalloc((void**)&box->positionMemory_dev, D * NP *sizeof(float));
        hipMalloc((void**)&box->grid_dev, M2 * box->EpM * sizeof(uint));

        //for parallel interactions
        IT = box->EpM * NG * NG;
        uint M_NG = box->M/NG + 0.9;
        hipMalloc((void**)&box->refGrid_dev, M_NG * M_NG * sizeof(uint));
        makeGridPattern2D(box);

        std::cout << "Made Box" << std::endl;
        
        return;
    }
    void killBox(Box* box){
        //for list
        hipFree(box->needUpdate_dev);
        hipFree(box->positionMemory_dev);
        hipFree(box->grid_dev);

        //for parallel interactions
        hipFree(box->refGrid_dev);

        //others
        killParticles(&box->p);
        std::cout << "Killed Box" << std::endl;

        return;
    }

    void prepareBox(Box* box){
        box->logFile << "Set InitPositions" << std::endl;
        scatterParticles(&box->p, box->L);
        hipMemcpy(box->p.diam, box->p.diam_dev, NP * sizeof(float),hipMemcpyDeviceToHost);
        for(uint n = 0; n < NP; n++){
            box->positionFile << box->p.diam[n] << " ";
        }
        box->positionFile << std::endl << std::endl;
        //set posMem and list
        updateGrid2D<<<NB,NT>>>(box, box->grid_dev, box->positionMemory_dev, box->p.x_dev);
        //remove overraps by using harmonic potential
        uint Nt = 20. / box->dt;
        /*for(int nt = 0; nt < Nt; nt++){
            tHarmonicDvlp();
            judgeUpdateGrid();
        }*/
        
        box->logFile << "-> SIP Done!" << std::endl;
        return;
    }
    void initBox(Box* box, uint ID){
        box->id = ID;
        std::cout << "Start Initialisation: ID = " << box->id << std::endl;
    
       //for record
        std::ostringstream positionFileName;
        positionFileName << "../pos/N" << (uint)NP << "/T" << Tfin << "/posBD_N" << (uint)NP << "_T" << Tfin << "_id" << box->id <<".data";
        box->positionFile.open(positionFileName.str().c_str());
        std::ostringstream logFileName;
        logFileName << "../log/N" << (uint)NP << "/T" << Tfin << "/logBD_N" << (uint)NP << "_T" << Tfin << "_id" << box->id <<".log";
        box->logFile.open(logFileName.str().c_str());

        box->logFile << "Start Initialisation: ID = " << box->id << std::endl;
        box->logFile << "Created Box ID = " << box->id << std::endl;
        
        //hotstart(Tinit >> 1)
        setdt_T(box, dt_INIT, Tinit);

        prepareBox(box);
    
        //equilibrateSys(30.0);
    
        //Tinit -> Tfin
        //coolSys(Tfin, tau);
    
        //setdt_T(dt_BD, Tfin);
    
        //equilibrateSys(10 * tau);
    
        box->logFile << "-> Init Done!" << std::endl;
        return;
    }
    //for grid
    __global__ void updateGrid2D(Box* box, uint* grid, float* positionMemory, float* x){
        uint n_global = blockIdx.x * blockDim.x + threadIdx.x;

        float bL = box->L;
        uint bM = box->M;
        uint bEpM = box->EpM;
        float rc = bL/(float)bM;
    
        uint gridPos[D];
        uint gridAddress;//[0, M * M - 1]
        uint n_m;
        uint counter;
    
        for(uint n = n_global; n < NP; n += NB* NT){
            gridPos[0] = (uint)x[n]/rc;
            gridPos[1] = (uint)x[NP+n]/rc;
            gridAddress = gridPos[1] * bM + gridPos[0];
            n_m = gridAddress * bEpM;
            counter = 1 + atomicAdd(&grid[n_m], 1);
            grid[n_m + counter] = n;
            positionMemory[n] = x[n];
            positionMemory[NP + n] = x[NP + n];
        }
    }

    __global__ void checkGrid(uint* needUpdate, float L, float* x, float* positionMemory){
        uint i_global = blockIdx.x * blockDim.x + threadIdx.x;
        uint i_local = threadIdx.x;
    
        __shared__ uint update[NT];
        update[i_local] = 0;
    
        float Lh = 0.5*L;
        float dx2;
        const float delta_x2 = a0 * a0 / D;
    
        for(uint i = i_global; i < D * N; i += NB * NT){
            dx2 = x[i] - positionMemory[i];
            if(dx2 > Lh){
                dx2 -= L;
            }
            if(dx2 < -Lh){
                dx2 += L;
            }
            dx2 *= dx2;
            if(dx2 > delta_x2){
                update[i_local] = 1;
            }
        }
        __syncthreads();
        //only for i_local = 0
        if(!i_local){
            uint foo = 0;
            for(uint i = 0; i < NT; i++){
                foo += update[i];
            }
            atomicAdd(needUpdate, foo);
        }
    }
    void judgeUpdateGrid(Box* box){
    
        checkGrid<<<NB,NT>>>(box->needUpdate_dev, box->L, box->p.x_dev, box->positionMemory);
        uint needUpdate;
        hipMemcpy(&needUpdate, box->needUpdate_dev, sizeof(uint), hipMemcpyDeviceToHost);
        if(needUpdate){
            updateGrid2D<<<NB,NT>>>(box, box->grid_dev, box->positionMemory_dev, box->p.x_dev);
        }
    }
}