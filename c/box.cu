#include "hip/hip_runtime.h"
#include "../h/box.cuh"

namespace PhysPeach{
    //setters and getters
    void makeBox(Box* box){

        makeParticles(&box->p);

        //default settings
        box->id = 0;
        box->dt = dt_MD;
        box->T = Tfin;
        box->L = sqrt((double)NP/(double)DNSTY);
        box->thermalFuctor = sqrt(2*box->T/box->dt);
        
        box->LDDir = "/LD";
        box->MDDir = "/MD";
        box->EDir = "/E";
        box->posDir = "/pos";
        box->velDir = "/vel";

        //Make dir tree
        std::ostringstream trajName;
        struct stat st;
        trajName << "../traj/N" << (uint)NP;
        if(stat(trajName.str().c_str(), &st) != 0){
            mkdir(trajName.str().c_str(), 0755);
            std::cout << "created " << trajName.str() << std::endl;
        }
        trajName << "/T" << Tfin;
        box->NTDir = trajName.str();
        if(stat(box->NTDir.c_str(), &st) != 0){
            mkdir(box->NTDir.c_str(), 0755);
            std::cout << "created " << box->NTDir << std::endl;

            mkdir((box->NTDir + box->LDDir).c_str(), 0755);
            mkdir((box->NTDir + box->LDDir + box->EDir).c_str(), 0755);
            mkdir((box->NTDir + box->LDDir + box->posDir).c_str(), 0755);
            mkdir((box->NTDir + box->LDDir + box->velDir).c_str(), 0755);
            
            mkdir((box->NTDir + box->MDDir).c_str(), 0755);
            mkdir((box->NTDir + box->MDDir + box->EDir).c_str(), 0755);
            mkdir((box->NTDir + box->MDDir + box->posDir).c_str(), 0755);
            mkdir((box->NTDir + box->MDDir + box->velDir).c_str(), 0755);
        }

        makeGrid(&box->g, box->L);

        std::cout << "Made Box" << std::endl;
        
        return;
    }

    void killBox(Box* box){

        //others
        killGrid(&box->g);
        killParticles(&box->p);
        std::cout << "Killed Box" << std::endl;

        return;
    }

    void prepareBox(Box* box){
        std::cout << "Set InitPositions" << std::endl;
        scatterParticles(&box->p, box->L);
        struct stat st;
        std::ostringstream diamName;
        diamName << "../traj/N" << NP << "/diam.data";
        if(stat(diamName.str().c_str(), &st) != 0){
            std::ofstream diamFile;
            diamFile.open(diamName.str().c_str());
            std::cout << "created " << diamName.str() << std::endl;
            hipMemcpy(box->p.diam, box->p.diam_dev, NP * sizeof(float),hipMemcpyDeviceToHost);
            for(uint n = 0; n < NP; n++){
                diamFile << box->p.diam[n] << std::endl;
            }
            diamFile.close();
        }
        //set posMem and list
        setIntVecZero<<<NB,NT>>>(box->g.cell_dev, box->g.M * box->g.M * box->g.EpM);
        updateGrid2D<<<NB,NT>>>(box->g, box->g.cell_dev, box->p.x_dev);
        //remove overraps by using harmonic potential
        uint Nt = 10. / box->dt;
        for(int nt = 0; nt < Nt; nt++){
            harmonicEvoBox(box);
        }
        
        std::cout << "-> SIP Done!" << std::endl;
        return;
    }
    void initBox(Box* box, uint ID){
        box->id = ID;
        std::cout << "Start Initialisation: ID = " << box->id << std::endl;
    
        //for record
        setdt_T(box, dt_INIT, Tfin);
        prepareBox(box);
        setdt_T(box, dt_BD, Tfin);
        equilibrateBox(box, tmax);
        std::cout << "-> Init Done!" << std::endl;
        return;
    }

    //time evolution
    inline void harmonicEvoBox(Box* box){
        culcHarmonicFint2D<<<NB,NT>>>(
            box->g, 
            box->g.refCell_dev, 
            box->g.cell_dev, 
            box->p.force_dev, 
            box->p.diam_dev, 
            box->p.x_dev
        );
        vEvoBD<<<NB,NT>>>(box->p.v_dev, box->dt, 0, box->p.force_dev, box->p.rndState_dev);
        removevg2D(&box->p);
        xEvo<<<NB,NT>>>(box->p.x_dev, box->dt, box->L, box->p.v_dev);
        checkUpdate(&box->g, box->dt, box->p.x_dev, box->p.v_dev);

        return;
    }
    inline void tEvoBox(Box* box){
        culcFint2D<<<IB,IT>>>(
            box->g, 
            box->g.refCell_dev, 
            box->g.cell_dev, 
            box->p.force_dev, 
            box->p.diam_dev, 
            box->p.x_dev
        );
        vEvoBD<<<NB,NT>>>(box->p.v_dev, box->dt, box->thermalFuctor, box->p.force_dev, box->p.rndState_dev);
        removevg2D(&box->p);
        xEvo<<<NB,NT>>>(box->p.x_dev, box->dt, box->L, box->p.v_dev);
        checkUpdate(&box->g, box->dt, box->p.x_dev, box->p.v_dev);
        
        return;
    }

    //equilibrations
    void equilibrateBox(Box* box, double teq){
        std::cout << "Equilibrate the System: ID = " << box->id << std::endl;
        uint Nt = teq/box->dt;
	    for (uint nt = 0; nt < Nt; nt++) {
		    tEvoBox(box);
	    }
	    std::cout << " -> Edone"<< box->id << std::endl;
        return;
    }

    //record
    void recPos(std::ofstream *of, Box* box){
        hipMemcpy(box->p.x, box->p.x_dev, D * NP * sizeof(double), hipMemcpyDeviceToHost);
	    for (int n = 0; n < NP; n++) {
		    for (char d = 0; d < D; d++) {
			    *of << box->p.x[d*NP+n] << " ";
            }
        }
        *of << std::endl;
        return;
    }
    void getData(Box* box){
        std::cout << "Starting time loop: ID = " << box->id << std::endl;
        uint Nt, tag;
        std::ofstream tFile;
        std::ofstream eFile;
        std::ofstream posFile;

        if(box->id == 1){
            std::cout << "getting liniarPlot datas in 5 secs" << std::endl;

            std::string tLinpltName = "/tliniar.data";
            tFile.open((box->NTDir + box->LDDir + tLinpltName).c_str());

            std::ostringstream eLinpltName;
            eLinpltName << box->NTDir + box->LDDir + box->EDir << "/liniar.data";
            eFile.open(eLinpltName.str().c_str());

            std::ostringstream posLinpltName;
            posLinpltName << box->NTDir + box->LDDir + box->posDir << "/liniar.data";
            posFile.open(posLinpltName.str().c_str());

            Nt = 5./box->dt;
            tag = 0;
            for(uint nt = 0; nt < Nt; nt++){
                tEvoBox(box);
                if(nt >= tag){
                    if(box->id == 1){
                        tFile << nt * box->dt << std::endl;
                    }
                    eFile << K(&box->p) << " " << U(&box->g, box->p.diam_dev, box->p.x_dev) << std::endl;
                    recPos(&posFile, box);
                    tag += 0.1/box->dt;
                }
            }
            posFile.close();
            eFile.close();
            tFile.close();
            std::string tLogpltName = "/tlog.data";
            tFile.open((box->NTDir + box->LDDir + tLogpltName).c_str());
        }

        std::cout << "getting logPlot datas" << std::endl;
        std::ostringstream eLogpltName;
        eLogpltName << box->NTDir + box->LDDir + box->EDir << "/id" << box->id << ".data";
        eFile.open(eLogpltName.str().c_str());

        std::ostringstream posLogpltName;
        posLogpltName << box->NTDir + box->LDDir + box->posDir << "/id" << box->id << ".data";
        posFile.open(posLogpltName.str().c_str());

        Nt = tmax/box->dt;
        tag = 10;
        for(uint nt = 0; nt <= Nt; nt++){
            tEvoBox(box);
            if(nt >= tag){
                if(box->id == 1){
                    tFile << nt * box->dt << std::endl;
                }
                eFile << K(&box->p) << " " << U(&box->g, box->p.diam_dev, box->p.x_dev) << std::endl;
                recPos(&posFile, box);
                tag *= 1.3;
            }
        }
        if(box->id == 1){
            tFile.close();
        }
        eFile.close();
        posFile.close();
        std::cout << "Every steps have been done: ID = " << box->id << std::endl << std::endl;
        return;
    }
    void benchmark(Box* box, uint loop){
        for(uint l = 0; l <=loop; l++){
            tEvoBox(box);
        }
        return;
    }
}