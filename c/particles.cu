#include "hip/hip_runtime.h"
#include "../h/particles.cuh"

namespace PhysPeach{
    __global__ void init_genrand_kernel(unsigned long long seed, hiprandState* state){
        uint i_global = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, i_global,0,&state[i_global]);
    }
    
    void makeParticles(Particles* p){
        //malloc host
        p->diam = (float*)malloc(NP * sizeof(float));
        p->x = (float*)malloc(D * NP*sizeof(float));
        p->v = (float*)malloc(D * NP * sizeof(float));
    
        //malloc device
        hipMalloc((void**)&p->diam_dev, NP * sizeof(float));
        hipMalloc((void**)&p->x_dev, D * NP * sizeof(float));
        hipMalloc((void**)&p->v_dev, D * NP * sizeof(float));
        hipMalloc((void**)&p->rndState_dev, D * NP *sizeof(hiprandState));
        hipMalloc((void**)&p->force_dev, D * NP*sizeof(float));
        
        //for setters and getters 
        hipMalloc((void**)&p->getNK_dev[0], D * NP * sizeof(float));
        hipMalloc((void**)&p->getNK_dev[1], D * NP * sizeof(float));
        hipMalloc((void**)&p->getNU_dev[0], NP * sizeof(float));
        hipMalloc((void**)&p->getNU_dev[1], NP * sizeof(float));
        hipMalloc((void**)&p->getNvg_dev, D * NP * sizeof(float));
        hipMalloc((void**)&p->Nvg_dev, D * sizeof(float)); 

        //set rnd seed
        init_genrand_kernel<<<NB,NT>>>((unsigned long long)genrand_int32(),p->rndState_dev);
        return;
    }
    
    void killParticles(Particles* p){
        free(p->diam);
        free(p->x);
        free(p->v);
        
        hipFree(p->diam_dev);
        hipFree(p->x_dev);
        hipFree(p->v_dev);
        hipFree(p->rndState_dev);
        hipFree(p->force_dev);

        hipFree(p->getNK_dev[0]);
        hipFree(p->getNK_dev[1]);
        hipFree(p->getNU_dev[0]);
        hipFree(p->getNU_dev[1]);
        hipFree(p->getNvg_dev);
        hipFree(p->Nvg_dev);
        return;
    }

    //setters and getters
    __global__ void setRndParticleStates(float l,float *diam, float *x, float *v ,hiprandState *rndState){
        uint i_global = blockIdx.x * blockDim.x + threadIdx.x;
        
        float atmp = a2 - a1;
        for(uint i = i_global; i < NP; i+=NB*NT){
            diam[i] = a1 + atmp * (i%2);
        }

        hiprandState localState = rndState[i_global];
        for(uint i = i_global; i < D * NP; i+=NB*NT){
            x[i] = l * hiprand_uniform(&localState);
            v[i] = 0.0;
        }
        rndState[i_global] = localState;
    }
    void scatterParticles(Particles* p, float L){
        //set positions by uniform random destribution
        setRndParticleStates<<<NB,NT>>>(L, p->diam_dev, p->x_dev, p->v_dev, p->rndState_dev);
        checkPeriodic<<<NB,NT>>>(L, p->x_dev);
    }
    __global__ void checkPeriodic(float L, float *x){
        uint n_global = blockIdx.x * blockDim.x + threadIdx.x;

        for(uint n = n_global; n < NP; n += NB*NT){
            if(x[n] > L){
                x[n] -= L;
            }
            else if (x[n] < 0){
                x[n] += L;
            }
        }
    }
}