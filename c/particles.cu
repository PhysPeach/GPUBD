#include "hip/hip_runtime.h"
#include "../h/particles.cuh"

namespace PhysPeach{
    __global__ void init_genrand_kernel(unsigned long long seed, hiprandState* state){
        unsigned int i_global = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, i_global,0,&state[i_global]);
    }
    
    void makeParticles(Particles* p){
        //malloc host
        p->diam = (float*)malloc(N*sizeof(float));
        p->x = (float*)malloc(D*N*sizeof(float));
        p->v = (float*)malloc(D*N*sizeof(float));
    
        //malloc device
        hipMalloc((void**)&p->diam_dev, N*sizeof(float));
        hipMalloc((void**)&p->x_dev, D*N*sizeof(float));
        hipMalloc((void**)&p->v_dev, D*N*sizeof(float));
        hipMalloc((void**)&p->rndState_dev, D*N*sizeof(hiprandState));
        hipMalloc((void**)&p->force_dev, D*N*sizeof(float));
    
        //set rnd seed
        init_genrand_kernel<<<NB,NT>>>((unsigned long long)genrand_int32(),p->rndState_dev);
        return;
    }
    
    void killParticles(Particles* p){
        free(p->diam);
        free(p->x);
        free(p->v);
        
        hipFree(p->diam_dev);
        hipFree(p->x_dev);
        hipFree(p->v_dev);
        hipFree(p->rndState_dev);
        hipFree(p->force_dev);
        return;
    }
}