#include "hip/hip_runtime.h"
#include "../h/particles.cuh"

namespace PhysPeach{
    __global__ void init_genrand_kernel(unsigned long long seed, hiprandState* state){
        unsigned int i_global = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, i_global,0,&state[i_global]);
    }
    
    void makeParticles(Particles* p){
        //malloc host
        p->diam = (float*)malloc(N*sizeof(float));
        p->x = (float*)malloc(D*N*sizeof(float));
        p->v = (float*)malloc(D*N*sizeof(float));
    
        //malloc device
        hipMalloc((void**)&p->diam_dev, N*sizeof(float));
        hipMalloc((void**)&p->x_dev, D*N*sizeof(float));
        hipMalloc((void**)&p->v_dev, D*N*sizeof(float));
        hipMalloc((void**)&p->rndState_dev, D*N*sizeof(hiprandState));
        hipMalloc((void**)&p->force_dev, D*N*sizeof(float));
    
        //set rnd seed
        init_genrand_kernel<<<NB,NT>>>((unsigned long long)genrand_int32(),p->rndState_dev);
        return;
    }
    
    void killParticles(Particles* p){
        free(p->diam);
        free(p->x);
        free(p->v);
        
        hipFree(p->diam_dev);
        hipFree(p->x_dev);
        hipFree(p->v_dev);
        hipFree(p->rndState_dev);
        hipFree(p->force_dev);
        return;
    }

    //setters and getters
    __global__ void setRndPositions(float l,float *diam_dev, float *x_dev, float *v_dev ,hiprandState *rndState_dev){
        unsigned int i_global = blockIdx.x * blockDim.x + threadIdx.x;
        
        float atmp = a2 - a1;
        for(unsigned int i = i_global; i < N; i+=NB*NT){
            diam_dev[i] = a1 + atmp * (i%2);
        }

        hiprandState localState = rndState_dev[i_global];
        for(unsigned int i = i_global; i < D * N; i+=NB*NT){
            x_dev[i] = l * hiprand_uniform(&localState);
            v_dev[i] = 0.0;
        }
        rndState_dev[i_global] = localState;
    }

    void initParticles(Particles* p, float L){
        //avoiding super overraps
        float Ltmp = L - 0.5 * (a1+a2);

        //set positions by uniform random destribution
        setRndPositions<<<NB,NT>>>(Ltmp, p->diam_dev, p->x_dev, p->v_dev, p->rndState_dev);
    }
}